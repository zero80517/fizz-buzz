#include <stdio.h>

#include <hip/hip_runtime.h>


#define BLOCK_SIZE 512

// no null terminator
__device__ void int2str(int value, char *buf, int size) {
  int i = size - 1;
  do {
    buf[i--] = (value % 10) + '0';
    value = value / 10;
  } while (value > 0);
}

// return str = "1       \n2       \nFizz    \n...\n\0"
__global__ void kernel_fizzbuzz(char *str, int allign, int n) {
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  int i = id + 1;
  int shift = 0;

  if (id < n)
  {
    if (i % 3 == 0) {
      str[id * allign] = 'F';
      str[id * allign + 1] = 'i';
      str[id * allign + 2] ='z';
      str[id * allign + 3] ='z';
      shift += 4;
    }
    if (i % 5 == 0) {
      str[id * allign + shift] = 'B';
      str[id * allign + shift + 1] = 'u';
      str[id * allign + shift + 2] ='z';
      str[id * allign + shift + 3] ='z';
      shift += 4;
    }
    if (!(i % 3 == 0 || i % 5 == 0)) {
      shift = floorf(log10f((float)i)) + 1;
      int2str(i, &str[id * allign], shift);
    }
    for (int s = shift; s < allign - 1; s++)
    {
      str[id * allign + s] = ' ';
    }
    str[id * allign + allign - 1] = '\n';
  }
}

int main() {
  int n = 100, allign = 9;
  char *host_str, *dev_str;
  host_str = (char *)malloc(n * allign * sizeof(char) + 1);
  hipMalloc((void **)&dev_str, n * allign * sizeof(char) + 1);

  dim3 block(BLOCK_SIZE);
  dim3 grid(n / BLOCK_SIZE + (n % BLOCK_SIZE ? 1 : 0));

  kernel_fizzbuzz<<<grid, block>>>(dev_str, allign, n);

  hipMemcpy(host_str, dev_str, n * allign * sizeof(char) + 1, hipMemcpyDeviceToHost);
  puts(host_str);

  free(host_str);
  hipFree(dev_str);

  return EXIT_SUCCESS;
}